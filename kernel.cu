#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <time.h>
#include <iostream>
#include <vector>
#include <cfloat>

// STB_IMAGE
#include "stb/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb/stb_image_write.h"

// Classe vec3
#include "vec3.h"

// Classe per il raggio
class ray
{
public:
    __device__ ray() {}
    __device__ ray(const Vec3 &a, const Vec3 &b)
    {
        A = a;
        B = b;
    }
    __device__ Vec3 origin() const { return A; }
    __device__ Vec3 direction() const { return B; }
    __device__ Vec3 point_at_parameter(float t) const { return A + t * B; }

    Vec3 A;
    Vec3 B;
};

// Record per gli oggetti
struct hit_record
{
    float t;
    Vec3 p;
    Vec3 normal;
};

// Classe astratta di oggetti che possono essere colpiti
class hitable
{
public:
    __device__ virtual bool hit(const ray &r, float t_min, float t_max, hit_record &rec) const = 0;
};

// Sfera (hitable)
class sphere : public hitable
{
public:
    __device__ sphere() {}
    __device__ sphere(Vec3 cen, float r) : center(cen), radius(r){};
    __device__ virtual bool hit(const ray &r, float tmin, float tmax, hit_record &rec) const;
    Vec3 center;
    float radius;
};

__device__ bool sphere::hit(const ray &r, float t_min, float t_max, hit_record &rec) const
{
    Vec3 oc = r.origin() - center;
    float a = dot(r.direction(), r.direction());
    float b = dot(oc, r.direction());
    float c = dot(oc, oc) - radius * radius;
    float discriminant = b * b - a * c;
    if (discriminant > 0)
    {
        float temp = (-b - sqrt(discriminant)) / a;
        if (temp < t_max && temp > t_min)
        {
            rec.t = temp;
            rec.p = r.point_at_parameter(rec.t);
            rec.normal = (rec.p - center) / radius;
            return true;
        }
        temp = (-b + sqrt(discriminant)) / a;
        if (temp < t_max && temp > t_min)
        {
            rec.t = temp;
            rec.p = r.point_at_parameter(rec.t);
            rec.normal = (rec.p - center) / radius;
            return true;
        }
    }
    return false;
};

// Lista di oggetti colpibili da Ray
class hitable_list : public hitable
{
public:
    __device__ hitable_list() {}
    __device__ hitable_list(hitable **l, int n)
    {
        list = l;
        list_size = n;
    }
    __device__ virtual bool hit(const ray &r, float tmin, float tmax, hit_record &rec) const;
    hitable **list;
    int list_size;
};

//
__device__ bool hitable_list::hit(const ray &r, float t_min, float t_max, hit_record &rec) const
{
    hit_record temp_rec;
    bool hit_anything = false;
    float closest_so_far = t_max;
    for (int i = 0; i < list_size; i++)
    {
        if (list[i]->hit(r, t_min, closest_so_far, temp_rec))
        {
            hit_anything = true;
            closest_so_far = temp_rec.t;
            rec = temp_rec;
        }
    }
    return hit_anything;
}

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line)
{
    if (result)
    {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " << file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__ bool hit_sphere(const Vec3 &center, float radius, const ray &r)
{
    Vec3 oc = r.origin() - center;
    float a = dot(r.direction(), r.direction());
    float b = 2.0f * dot(oc, r.direction());
    float c = dot(oc, oc) - radius * radius;
    float discriminant = b * b - 4.0f * a * c;
    return (discriminant > 0.0f);
}

__device__ Vec3 color(const ray &r, hitable **world)
{
    hit_record rec;
    if ((*world)->hit(r, 0.0, FLT_MAX, rec))
    {
        return 0.5f * Vec3(rec.normal.x() + 1.0f, rec.normal.y() + 1.0f, rec.normal.z() + 1.0f);
    }
    else
    {
        Vec3 unit_direction = unit_vector(r.direction());
        float t = 0.5f * (unit_direction.y() + 1.0f);
        return (1.0f - t) * Vec3(1.0, 1.0, 1.0) + t * Vec3(0.5, 0.7, 1.0);
    }
}

__global__ void render(Vec3 *fb, int max_x, int max_y,
                       Vec3 lower_left_corner, Vec3 horizontal,
                       Vec3 vertical, Vec3 origin, hitable **world)
{
    int const i = threadIdx.x + blockIdx.x * blockDim.x; // Mi identifica i thread sulle ascisse della griglia
    int const j = threadIdx.y + blockIdx.y * blockDim.y; // Mi identifica i thread sulle ordinate della griglia

    if ((i >= max_x) || (j >= max_y))
        return;

    // Indice del pixel su memoria contigua
    int const pixel_index = j * max_x + i;

    float u = float(i) / float(max_x);
    float v = float(j) / float(max_y);

    // Costruzione del raggio
    ray r(origin, lower_left_corner + u * horizontal + v * vertical);
    fb[pixel_index] = color(r, world);
}

__global__ void create_world(hitable **d_list, hitable **d_world)
{
    // Ci assicuriamo che il popolamento di entrambe le liste avvenga soltanto una volta
    if (threadIdx.x == 0 && blockIdx.x == 0)
    {
        *(d_list) = new sphere(Vec3(0, 0, -1), 0.5);
        *(d_list + 1) = new sphere(Vec3(0, -100.5, -1), 100);
        *d_world = new hitable_list(d_list, 2);
    }
}

__global__ void free_world(hitable **d_list, hitable **d_world)
{
    delete *(d_list);
    delete *(d_list + 1);
    delete *d_world;
}

int main(void)
{
    int constexpr width = 1200;
    int constexpr height = 600;
    int constexpr tx = 8;
    int constexpr ty = 8;

    std::vector<uint8_t> image;

    std::cerr << "Rendering a " << width << "x" << height << " image ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = width * height;
    size_t fb_size = num_pixels * sizeof(Vec3);

    // allocate FB
    Vec3 *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    // Allocazione sulla GPU della lista di hitables
    hitable **d_list;
    checkCudaErrors(hipMalloc((void **)&d_list, 2 * sizeof(hitable *)));
    // Allocazione del mondo che conterrà gli oggetti
    hitable **d_world;
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hitable *)));
    // Popolamento da device di entrambe le liste
    create_world<<<1, 1>>>(d_list, d_world);

    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Render dell'immagine + timing
    dim3 blocks(width / tx + 1, height / ty + 1);
    dim3 threads(tx, ty);

    //   clock_t start, stop;
    // start = clock();
    // Il mondo (d_world) viene passato come parametro alla funzione di rendering
    render<<<blocks, threads>>>(fb, width, height,
                                Vec3(-2.0, -1.0, -1.0),
                                Vec3(4.0, 0.0, 0.0),
                                Vec3(0.0, 2.0, 0.0),
                                Vec3(0.0, 0.0, 0.0),
                                d_world);

    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    //double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    //std::cerr << "took " << timer_seconds << " seconds.\n";

    // Salvo l'immagine (host code)
    std::cout << "P3\n"
              << width << " " << height << "\n255\n";
    for (int j = height - 1; j >= 0; j--)
    {
        for (int i = 0; i < width; i++)
        {
            size_t pixel_index = j * width + i;
            int const ir = int(255.99f * fb[pixel_index].r());
            int const ig = int(255.99f * fb[pixel_index].g());
            int const ib = int(255.99f * fb[pixel_index].b());

            image.push_back(ir);
            image.push_back(ig);
            image.push_back(ib);
        }
    }
    stbi_write_png("output.png", width, height, 3, image.data(), 0);

    // Si libera la device memory
    free_world<<<1, 1>>>(d_list, d_world);
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(fb));

    return 0;
}
